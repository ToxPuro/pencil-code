#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2021, Johannes Pekkila, Miikka Vaisala.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "acc_runtime.h"

#include <vector> // tbconfig

#include "errchk.h"
#include "math_utils.h"

#if AC_USE_HIP
#include <hip/hip_runtime.h> // Needed in files that include kernels
#endif

#define USE_COMPRESSIBLE_MEMORY (0)

#include "acc/implementation.h"

Volume
get_bpg(const Volume dims, const Volume tpb)
{
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: // Fallthrough
  case EXPLICIT_CACHING: // Fallthrough
  case EXPLICIT_CACHING_3D_BLOCKING: {
    return (Volume){
        (size_t)ceil(1. * dims.x / tpb.x),
        (size_t)ceil(1. * dims.y / tpb.y),
        (size_t)ceil(1. * dims.z / tpb.z),
    };
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_bpg");
    return (Volume){0, 0, 0};
  }
  }
}

bool
is_valid_configuration(const Volume dims, const Volume tpb)
{
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  const size_t warp_size = props.warpSize;
  const size_t xmax      = (size_t)(warp_size * ceil(1. * dims.x / warp_size));
  const size_t ymax      = (size_t)(warp_size * ceil(1. * dims.y / warp_size));
  const size_t zmax      = (size_t)(warp_size * ceil(1. * dims.z / warp_size));
  const bool too_large   = (tpb.x > xmax) || (tpb.y > ymax) || (tpb.z > zmax);

  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {

    if (too_large)
      return false;

    return true;
  }
  case EXPLICIT_CACHING: // Fallthrough
  case EXPLICIT_CACHING_3D_BLOCKING: {

    // For some reason does not work without this
    return !(dims.x % tpb.x) && !(dims.y % tpb.y) && !(dims.z % tpb.z);
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in is_valid_configuration");
    return false;
  }
  }
}

size_t
get_smem(const Volume tpb, const size_t stencil_order,
         const size_t bytes_per_elem)
{
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {
    return 0;
  }
  case EXPLICIT_CACHING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) * tpb.z *
           bytes_per_elem;
  }
  case EXPLICIT_CACHING_3D_BLOCKING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           (tpb.z + stencil_order) * bytes_per_elem;
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_smem");
    return (size_t)-1;
  }
  }
}

/*
// Device info (TODO GENERIC)
// Use the maximum available reg count per thread
#define REGISTERS_PER_THREAD (255)
#define MAX_REGISTERS_PER_BLOCK (65536)
#if AC_DOUBLE_PRECISION
#define MAX_THREADS_PER_BLOCK                                                  \
  (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD / 2)
#else
#define MAX_THREADS_PER_BLOCK (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD)
#endif
*/

__device__ __constant__ AcMeshInfo d_mesh_info;

// Astaroth 2.0 backwards compatibility START
#define d_multigpu_offset (d_mesh_info.int3_params[AC_multigpu_offset])

int __device__ __forceinline__
DCONST(const AcIntParam param)
{
  return d_mesh_info.int_params[param];
}
int3 __device__ __forceinline__
DCONST(const AcInt3Param param)
{
  return d_mesh_info.int3_params[param];
}
AcReal __device__ __forceinline__
DCONST(const AcRealParam param)
{
  return d_mesh_info.real_params[param];
}
AcReal3 __device__ __forceinline__
DCONST(const AcReal3Param param)
{
  return d_mesh_info.real3_params[param];
}

#define DEVICE_VTXBUF_IDX(i, j, k)                                             \
  ((i) + (j)*DCONST(AC_mx) + (k)*DCONST(AC_mxy))

__device__ constexpr int
IDX(const int i)
{
  return i;
}

#if 1
__device__ __forceinline__ int
IDX(const int i, const int j, const int k)
{
  return DEVICE_VTXBUF_IDX(i, j, k);
}
#else
constexpr __device__ int
IDX(const uint i, const uint j, const uint k)
{
  /*
  const int precision   = 32; // Bits
  const int dimensions  = 3;
  const int bits = ceil(precision / dimensions);
  */
  const int dimensions = 3;
  const int bits       = 11;

  uint idx = 0;
#pragma unroll
  for (uint bit = 0; bit < bits; ++bit) {
    const uint mask = 0b1 << bit;
    idx |= ((i & mask) << 0) << (dimensions - 1) * bit;
    idx |= ((j & mask) << 1) << (dimensions - 1) * bit;
    idx |= ((k & mask) << 2) << (dimensions - 1) * bit;
  }
  return idx;
}
#endif

// Only used in reductions
__device__ __forceinline__ int
IDX(const int3 idx)
{
  return DEVICE_VTXBUF_IDX(idx.x, idx.y, idx.z);
}

#define Field3(x, y, z) make_int3((x), (y), (z))
#define print printf                          // TODO is this a good idea?
#define len(arr) sizeof(arr) / sizeof(arr[0]) // Leads to bugs if the user
// passes an array into a device function and then calls len (need to modify
// the compiler to always pass arrays to functions as references before
// re-enabling)

#include "random.cuh"

#include "user_kernels.h"

typedef struct {
  Kernel kernel;
  int3 dims;
  dim3 tpb;
} TBConfig;

static std::vector<TBConfig> tbconfigs;

static TBConfig getOptimalTBConfig(const Kernel kernel, const int3 dims,
                                   VertexBufferArray vba);

static __global__ void
flush_kernel(AcReal* arr, const size_t n, const AcReal value)
{
  const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n)
    arr[idx] = value;
}

AcResult
acKernelFlush(AcReal* arr, const size_t n, const AcReal value)
{
  const size_t tpb = 256;
  const size_t bpg = (size_t)(ceil((double)n / tpb));
  flush_kernel<<<bpg, tpb>>>(arr, n, value);
  ERRCHK_CUDA_KERNEL_ALWAYS();
  return AC_SUCCESS;
}

#if USE_COMPRESSIBLE_MEMORY
#include <hip/hip_runtime.h>

#define ERRCHK_CU_ALWAYS(x) ERRCHK_ALWAYS((x) == hipSuccess)

static hipError_t
mallocCompressible(void** addr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity;
  ERRCHK_CU_ALWAYS(hipMemGetAllocationGranularity(
      &granularity, &prop, hipMemAllocationGranularityMinimum));

  // Pad to align
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  hipDeviceptr_t dptr;
  ERRCHK_ALWAYS(hipMemAddressReserve(&dptr, bytes, 0, 0, 0) == hipSuccess);

  hipMemGenericAllocationHandle_t handle;
  ERRCHK_ALWAYS(hipMemCreate(&handle, bytes, &prop, 0) == hipSuccess)

  // Check if hipMemCreate was able to allocate compressible memory.
  hipMemAllocationProp alloc_prop;
  memset(&alloc_prop, 0, sizeof(hipMemAllocationProp));
  hipMemGetAllocationPropertiesFromHandle(&alloc_prop, handle);
  ERRCHK_ALWAYS(alloc_prop.allocFlags.compressionType ==
                CU_MEM_ALLOCATION_COMP_GENERIC);

  ERRCHK_ALWAYS(hipMemMap(dptr, bytes, 0, handle, 0) == hipSuccess);
  ERRCHK_ALWAYS(hipMemRelease(handle) == hipSuccess);

  hipMemAccessDesc accessDescriptor;
  accessDescriptor.location.id   = prop.location.id;
  accessDescriptor.location.type = prop.location.type;
  accessDescriptor.flags         = hipMemAccessFlagsProtReadWrite;

  ERRCHK_ALWAYS(hipMemSetAccess(dptr, bytes, &accessDescriptor, 1) ==
                hipSuccess);

  *addr = (void*)dptr;
  return hipSuccess;
}

static void
freeCompressible(void* ptr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity = 0;
  ERRCHK_ALWAYS(hipMemGetAllocationGranularity(
                    &granularity, &prop, hipMemAllocationGranularityMinimum) ==
                hipSuccess);
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  ERRCHK_ALWAYS(ptr);
  ERRCHK_ALWAYS(hipMemUnmap((hipDeviceptr_t)ptr, bytes) == hipSuccess);
  ERRCHK_ALWAYS(hipMemAddressFree((hipDeviceptr_t)ptr, bytes) == hipSuccess);
}
#endif

VertexBufferArray
acVBACreate(const size_t count)
{
  VertexBufferArray vba;

  const size_t bytes = sizeof(vba.in[0][0]) * count;

//#define ADJACENT_VERTEX_BUFFERS 1
#if AC_ADJACENT_VERTEX_BUFFERS
  const size_t allbytes = bytes*NUM_VTXBUF_HANDLES;
  AcReal *allbuf_in, *allbuf_out;

  ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&allbuf_in, allbytes));
  ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&allbuf_out, allbytes));
  acKernelFlush(allbuf_in, count*NUM_VTXBUF_HANDLES, (AcReal)NAN);
  ERRCHK_CUDA_ALWAYS(hipMemset((void*)allbuf_out, 0, allbytes));

  vba.in[0]=allbuf_in; vba.out[0]=allbuf_out;
printf("i,vbas[0]= 0 %p \n",vba.in[0],vba.out[0]);
  for (size_t i = 1; i < NUM_VTXBUF_HANDLES; ++i) {
    vba.in [i]=vba.in [i-1]+count;
    vba.out[i]=vba.out[i-1]+count;
printf("i,vbas[i]= %d %p \n",i,vba.in[i],vba.out[i]);
  }
#else
  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
#if USE_COMPRESSIBLE_MEMORY
    ERRCHK_CUDA_ALWAYS(mallocCompressible((void**)&vba.in[i], bytes));
    ERRCHK_CUDA_ALWAYS(mallocCompressible((void**)&vba.out[i], bytes));
#else
    ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&vba.in[i], bytes));
    ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&vba.out[i], bytes));
#endif
printf("i,vbas[i]= %d %p %p\n",i,vba.in[i],vba.out[i]);

    // Set vba.in data to all-nan and vba.out to 0
    acKernelFlush(vba.in[i], count, (AcReal)NAN);
    acKernelFlush(vba.out[i], count, (AcReal)0.0);
  }
#endif
  return vba;
}

void
acVBADestroy(VertexBufferArray* vba)
{
  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
#if USE_COMPRESSIBLE_MEMORY
    freeCompressible(vba->in[i], vba->bytes);
    freeCompressible(vba->out[i], vba->bytes);
#else
    hipFree(vba->in[i]);
    hipFree(vba->out[i]);
#endif
    vba->in[i]  = NULL;
    vba->out[i] = NULL;
  }
  vba->bytes = 0;
}

AcResult
acLaunchKernel(Kernel kernel, const hipStream_t stream, const int3 start,
               const int3 end, VertexBufferArray vba)
{
  const int3 n = end - start;

  const dim3 tpb = getOptimalTBConfig(kernel, n, vba).tpb;
  ERRCHK(tpb.x*tpb.y*tpb.z<=1024);
  const dim3 bpg((unsigned int)ceil(n.x / double(tpb.x)), //
                 (unsigned int)ceil(n.y / double(tpb.y)), //
                 (unsigned int)ceil(n.z / double(tpb.z)));
  const size_t smem = 0;
//printf("before launch tpb,bpg=%d %d %d %d %d %d \n",tpb.x,tpb.y,tpb.z,bpg.x,bpg.y,bpg.z);
//printf("before launch start,end=%d %d %d %d %d %d \n",start.x,start.y,start.z,end.x,end.y,end.z);
  kernel<<<bpg, tpb, smem, stream>>>(start, end, vba);
  ERRCHK_CUDA_KERNEL();

  return AC_SUCCESS;
}

AcResult
acLoadStencil(const Stencil stencil, const hipStream_t stream,
              const AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyToSymbolAsync(HIP_SYMBOL(
      stencils), data, bytes, stencil * bytes, hipMemcpyHostToDevice, stream);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

AcResult
acStoreStencil(const Stencil stencil, const hipStream_t stream,
               AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyFromSymbolAsync(
      data, HIP_SYMBOL(stencils), bytes, stencil * bytes, hipMemcpyDeviceToHost, stream);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

#define GEN_LOAD_UNIFORM(LABEL_UPPER, LABEL_LOWER)                             \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
                                                                               \
  const size_t offset = (size_t)&d_mesh_info.LABEL_LOWER##_params[param] -     \
                        (size_t)&d_mesh_info;                                  \
                                                                               \
  const hipError_t retval = hipMemcpyToSymbolAsync(HIP_SYMBOL(                          \
      d_mesh_info), &value, sizeof(value), offset, hipMemcpyHostToDevice,      \
      stream);                                                                 \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acLoadRealUniform(const hipStream_t stream, const AcRealParam param,
                  const AcReal value)
{
  if (isnan(value)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value %g to device constant %s. "
            "Skipping.\n",
            (double)value, realparam_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL, real);
}

AcResult
acLoadReal3Uniform(const hipStream_t stream, const AcReal3Param param,
                   const AcReal3 value)
{
  if (isnan(value.x) || isnan(value.y) || isnan(value.z)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value (%g, %g, %g) to device constant "
            "%s. Skipping.\n",
            (double)value.x, (double)value.y, (double)value.z,
            real3param_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL3, real3);
}

AcResult
acLoadIntUniform(const hipStream_t stream, const AcIntParam param,
                 const int value)
{
  GEN_LOAD_UNIFORM(INT, int);
}

AcResult
acLoadInt3Uniform(const hipStream_t stream, const AcInt3Param param,
                  const int3 value)
{
  GEN_LOAD_UNIFORM(INT3, int3);
}

#define GEN_STORE_UNIFORM(LABEL_UPPER, LABEL_LOWER)                            \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
                                                                               \
  const size_t offset = (size_t)&d_mesh_info.LABEL_LOWER##_params[param] -     \
                        (size_t)&d_mesh_info;                                  \
                                                                               \
  const hipError_t retval = hipMemcpyFromSymbolAsync(                        \
      value, HIP_SYMBOL(d_mesh_info), sizeof(*value), offset, hipMemcpyDeviceToHost,      \
      stream);                                                                 \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acStoreRealUniform(const hipStream_t stream, const AcRealParam param,
                   AcReal* value)
{
  GEN_STORE_UNIFORM(REAL, real);
}

AcResult
acStoreReal3Uniform(const hipStream_t stream, const AcReal3Param param,
                    AcReal3* value)
{
  GEN_STORE_UNIFORM(REAL3, real3);
}

AcResult
acStoreIntUniform(const hipStream_t stream, const AcIntParam param, int* value)
{
  GEN_STORE_UNIFORM(INT, int);
}

AcResult
acStoreInt3Uniform(const hipStream_t stream, const AcInt3Param param,
                   int3* value)
{
  GEN_STORE_UNIFORM(INT3, int3);
}

static TBConfig
autotune(const Kernel kernel, const int3 dims, VertexBufferArray vba)
{
  printf("Autotuning kernel %p, block (%d, %d, %d)... ", kernel, dims.x, dims.y,
         dims.z);
  fflush(stdout);
// suppress autotuning for the moment; blocksize seems to be limited to 256
  return (TBConfig){
    .kernel = kernel,
    .dims = dims,
    .tpb = (dim3){64,2,2}
  };

  TBConfig c = {
      .kernel = kernel,
      .dims   = dims,
      .tpb    = (dim3){0, 0, 0},
  };

  const int3 start = (int3){
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
  };
  const int3 end = start + dims;

  dim3 best_tpb(0, 0, 0);
  float best_time     = INFINITY;
  const int num_iters = 2;

  // Get device hardware information
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  const int max_threads_per_block = MAX_THREADS_PER_BLOCK
                                        ? min(props.maxThreadsPerBlock,
                                              MAX_THREADS_PER_BLOCK)
                                        : props.maxThreadsPerBlock;
  const size_t max_smem           = props.sharedMemPerBlock;

  for (int z = 1; z <= max_threads_per_block; ++z) {
    for (int y = 1; y <= max_threads_per_block; ++y) {
      for (int x = max(y, z); x <= max_threads_per_block; ++x) {

        if (x * y * z > max_threads_per_block)
          break;

        // if (x * y * z * max_regs_per_thread > max_regs_per_block)
        //  break;

        // if (max_regs_per_block / (x * y * z) < min_regs_per_thread)
        //   continue;

        // if (x < y || x < z)
        //   continue;

        const dim3 tpb(x, y, z);
        const dim3 bpg    = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));
        const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER,
                                     sizeof(AcReal));

        if (smem > max_smem)
          continue;

        if ((x * y * z) % props.warpSize)
          continue;

        if (!is_valid_configuration(to_volume(dims), to_volume(tpb)))
          continue;

#if VECTORIZED_LOADS
        const size_t window = tpb.x + STENCIL_ORDER;

        // Vectorization criterion
        if (window % veclen) // Window not divisible into vectorized blocks
          continue;

        if (dims.x % tpb.x)
          continue;

          // May be too strict
          // if (dims.x % tpb.x || dims.y % tpb.y || dims.z % tpb.z)
          //   continue;
#endif
#if 0 // Disabled for now (waiting for cleanup)
#if USE_SMEM
        const size_t max_smem = 128 * 1024;
        if (smem > max_smem)
          continue;

#if VECTORIZED_LOADS
        const size_t window = tpb.x + STENCIL_ORDER;

        // Vectorization criterion
        if (window % veclen) // Window not divisible into vectorized blocks
          continue;

        if (dims.x % tpb.x || dims.y % tpb.y || dims.z % tpb.z)
          continue;
#endif

          //  Padding criterion
          //  TODO (cannot be checked here)
#else
        if ((x * y * z) % warp_size)
          continue;
#endif
#endif

        // printf("%d, %d, %d: %lu\n", tpb.x, tpb.y, tpb.z, smem);

        hipEvent_t tstart, tstop;
        hipEventCreate(&tstart);
        hipEventCreate(&tstop);

        hipDeviceSynchronize();
        hipEventRecord(tstart); // Timing start
        for (int i = 0; i < num_iters; ++i)
          kernel<<<bpg, tpb, smem>>>(start, end, vba);
        hipEventRecord(tstop); // Timing stop
        hipEventSynchronize(tstop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, tstart, tstop);

        hipEventDestroy(tstart);
        hipEventDestroy(tstop);

        // Discard failed runs (attempt to clear the error to hipSuccess)
        if (hipGetLastError() != hipSuccess) {
          // Exit in case of unrecoverable error that needs a device reset
          ERRCHK_CUDA_KERNEL_ALWAYS();
          ERRCHK_CUDA_ALWAYS(hipGetLastError());
          continue;
        }

        if (milliseconds < best_time) {
          best_time = milliseconds;
          best_tpb  = tpb;
        }

        // printf("Auto-optimizing... Current tpb: (%d, %d, %d), time %f ms\n",
        //        tpb.x, tpb.y, tpb.z, (double)milliseconds / num_iters);
        // fflush(stdout);
      }
    }
  }
  c.tpb = best_tpb;

  // printf("\tThe best tpb: (%d, %d, %d), time %f ms\n", best_tpb.x,
  // best_tpb.y,
  //        best_tpb.z, (double)best_time / num_iters);

  FILE* fp = fopen("autotune.csv", "a");
  ERRCHK_ALWAYS(fp);
#if IMPLEMENTATION == SMEM_HIGH_OCCUPANCY_CT_CONST_TB
  fprintf(fp, "%d, (%d, %d, %d), (%d, %d, %d), %g\n", IMPLEMENTATION, nx, ny,
          nz, best_tpb.x, best_tpb.y, best_tpb.z,
          (double)best_time / num_iters);
#else
  fprintf(fp, "%d, %d, %d, %d, %d, %d, %d, %g\n", IMPLEMENTATION, dims.x,
          dims.y, dims.z, best_tpb.x, best_tpb.y, best_tpb.z,
          (double)best_time / num_iters);
#endif
  fclose(fp);

  if (c.tpb.x * c.tpb.y * c.tpb.z <= 0) {
    fprintf(stderr,
            "Fatal error: failed to find valid thread block dimensions.\n");
  }
  ERRCHK_ALWAYS(c.tpb.x * c.tpb.y * c.tpb.z > 0);
  return c;
}

static TBConfig
getOptimalTBConfig(const Kernel kernel, const int3 dims, VertexBufferArray vba)
{
  for (auto c : tbconfigs) {
    if (c.kernel == kernel && c.dims == dims)
      return c;
  }
  TBConfig c = autotune(kernel, dims, vba);
  tbconfigs.push_back(c);
  return c;
}
