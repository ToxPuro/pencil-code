#include "hip/hip_runtime.h"
/**
    Microbenchmark the GPU caches in 1D stencil computations and generate a plottable .csv output

    Examples:
        # Usage
        ./bwtest-benchmark <problem size in bytes> <working set size in bytes>

        # 256 MiB problem size and working set of size 8 (one double), i.e. halo r=0
        ./bwtest-benchmark 268435456 8

        # 3-point von Neumann stencil
        ./bwtest-benchmark 268435456 24

        # Profiling
        cmake -DUSE_HIP=ON .. &&\
        make -j &&\
        rocprof --trace-start off -i ~/rocprof-input-metrics.txt ./bwtest-benchmark 268435456 256

cat ~/rocprof-input-metrics.txt
```
# Perf counters group 1
pmc : Wavefronts VALUInsts SALUInsts SFetchInsts
# Perf counters group 2
pmc : TCC_HIT[0], TCC_MISS[0], TCC_HIT_sum, TCC_MISS_sum
# Perf counters group 3
pmc: L2CacheHit MemUnitBusy LDSBankConflict

# Filter by dispatches range, GPU index and kernel names
# supported range formats: "3:9", "3:", "3"
#range: 0 : 16
gpu: 0 1 2 3
#kernel: singlepass_solve
```
*/
#include <stdio.h>
#include <stdlib.h>

#if AC_USE_HIP
#include "hip.h"
#include <hip/hip_runtime.h> // Needed in files that include kernels
#include <roctracer_ext.h>   // Profiling
#else
#include <hip/hip_runtime_api.h> // Profiling
#include <hip/hip_runtime_api.h>  // hipStream_t
#endif

#include "common.h"

// #define USE_SMEM (0) // Set with cmake
// #define MAX_THREADS_PER_BLOCK (0) // Set with cmake

#if USE_SMEM
static size_t
get_smem(const int tpb, const int halo)
{
    return (tpb + 2 * halo) * sizeof(double);
}

__global__ void
#if MAX_THREADS_PER_BLOCK
__launch_bounds__(MAX_THREADS_PER_BLOCK)
#endif
    kernel(const int halo, const Array in, Array out)
{
    extern __shared__ double smem[];

    const int base_idx = blockIdx.x * blockDim.x;
    for (int sid = threadIdx.x; sid < (int)(blockDim.x + 2 * halo); sid += blockDim.x)
        if (sid + base_idx < in.count)
            smem[sid] = in.data[sid + base_idx];
    __syncthreads();

    const int tid = (int)(threadIdx.x + blockIdx.x * blockDim.x) + halo;
    if (tid < in.count - halo) {

        double tmp = 0.0;
        for (int i = 0; i < 2 * halo + 1; ++i)
            tmp += smem[threadIdx.x + i];

        out.data[tid] = tmp;
    }
}
#else
static size_t
get_smem(const int tpb, const int halo)
{
    (void)tpb;  // Unused
    (void)halo; // Unused
    return 0;
}

__global__ void
#if MAX_THREADS_PER_BLOCK
__launch_bounds__(MAX_THREADS_PER_BLOCK)
#endif
    kernel(const int halo, const Array in, Array out)
{
    const int tid = (int)(threadIdx.x + blockIdx.x * blockDim.x);

    if (halo <= tid && tid < (int)in.count - halo) {
        double tmp = 0.0;

        for (int i = -halo; i <= halo; ++i)
            tmp += in.data[tid + i];

        out.data[tid] = tmp;
    }
}
#endif

void
model_kernel(const int halo, const Array in, Array out)
{
    for (int tid = 0; tid < (int)in.count; ++tid) {
        if (halo <= tid && tid < (int)in.count - halo) {

            double tmp = 0.0;
            for (int i = -halo; i <= halo; ++i)
                tmp += in.data[tid + i];

            out.data[tid] = tmp;
        }
    }
}

typedef struct {
    size_t count;
    int halo;
    size_t tpb;
    size_t bpg;
    size_t smem;
} KernelConfig;

/** Returns the optimal threadblock dimensions for a given problem size */
static KernelConfig
autotune(const size_t count, const int halo)
{
    Array a = arrayCreate(count, true);
    Array b = arrayCreate(count, true);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    const size_t warp_size             = (size_t)props.warpSize;
    const size_t max_smem              = (size_t)props.sharedMemPerBlock;
    const size_t max_threads_per_block = MAX_THREADS_PER_BLOCK
                                             ? (size_t)min(props.maxThreadsPerBlock,
                                                           MAX_THREADS_PER_BLOCK)
                                             : (size_t)props.maxThreadsPerBlock;

    // Warmup
    hipEvent_t tstart, tstop;
    hipEventCreate(&tstart);
    hipEventCreate(&tstop);
    hipEventRecord(tstart); // Timing start
    for (size_t i = 0; i < 1; ++i)
        kernel<<<1, 1, max_smem>>>(halo, a, b);
    hipEventRecord(tstop); // Timing stop
    hipEventSynchronize(tstop);
    hipEventDestroy(tstart);
    hipEventDestroy(tstop);
    hipDeviceSynchronize();

    // Tune
    KernelConfig c  = {.count = count, .halo = halo, .tpb = 0, .bpg = 0, .smem = 0};
    float best_time = INFINITY;
    for (size_t tpb = 1; tpb <= max_threads_per_block; ++tpb) {

        if (tpb > max_threads_per_block)
            break;

        if (tpb % warp_size)
            continue;

        const size_t bpg  = (size_t)ceil(1. * count / tpb);
        const size_t smem = get_smem(tpb, halo);

        if (smem > max_smem)
            continue;

        printf("Current KernelConfig {.count = %lu, .halo = %d, .tpb = %lu, .bpg = %lu, .smem = "
               "%lu}",
               c.count, c.halo, tpb, bpg, smem);

        hipEventCreate(&tstart);
        hipEventCreate(&tstop);

        hipDeviceSynchronize();
        hipEventRecord(tstart); // Timing start
        for (int i = 0; i < 3; ++i)
            kernel<<<bpg, tpb, smem>>>(halo, a, b);
        hipEventRecord(tstop); // Timing stop
        hipEventSynchronize(tstop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, tstart, tstop);

        hipEventDestroy(tstart);
        hipEventDestroy(tstop);

        ERRCHK_CUDA_KERNEL_ALWAYS();
        //  Discard failed runs (attempt to clear the error to hipSuccess)
        if (hipGetLastError() != hipSuccess) {
            // Exit in case of unrecoverable error that needs a device reset
            if (hipGetLastError() != hipSuccess) {
                fprintf(stderr, "Unrecoverable CUDA error\n");
                exit(EXIT_FAILURE);
            }
            continue;
        }

        // printf("KernelConfig {.tpb = %lu, .bpg = %lu}\n", tpb, bpg);
        printf(", Time elapsed: %g ms\n", (double)milliseconds);
        if (milliseconds < best_time) {
            best_time = milliseconds;
            c.tpb     = tpb;
            c.bpg     = bpg;
            c.smem    = smem;
        }
    }
    printf("KernelConfig {.count = %lu, .halo = %d, .tpb = %lu, .bpg = %lu, .smem = %lu}\n",
           c.count, c.halo, c.tpb, c.bpg, c.smem);

    arrayDestroy(&a);
    arrayDestroy(&b);

#if USE_SMEM
    ERRCHK_ALWAYS(c.smem);
#endif

    return c;
}

void
verify(const KernelConfig c)
{
    const size_t count = c.count;
    const size_t tpb   = c.tpb;
    const size_t bpg   = c.bpg;
    const size_t smem  = c.smem;
    const int halo     = c.halo;

    Array ahost = arrayCreate(count, false);
    Array bhost = arrayCreate(count, false);
    Array a     = arrayCreate(count, true);
    Array b     = arrayCreate(count, true);

    arrayRandomize(&ahost);
    model_kernel(halo, ahost, bhost);

    const size_t bytes = count * sizeof(ahost.data[0]);
    hipMemcpy(a.data, ahost.data, bytes, hipMemcpyHostToDevice);
    kernel<<<bpg, tpb, smem>>>(halo, a, b);
    hipMemcpy(ahost.data, b.data, bytes, hipMemcpyDeviceToHost);

    const double* candidate = ahost.data;
    const double* model     = bhost.data;

    for (size_t i = halo; i < ahost.count - halo; ++i) {
        if (model[i] != candidate[i]) {
            fprintf(stderr, "Failure at %lu: %g (host) and %g (device)\n", i, model[i],
                    candidate[i]);
        }
    }

    arrayDestroy(&a);
    arrayDestroy(&b);
    arrayDestroy(&ahost);
    arrayDestroy(&bhost);

    printf("Results verified\n");
}

static void
benchmark(const KernelConfig c)
{
    const size_t num_iters = 5;

    // Allocate
    Array a = arrayCreate(c.count, true);
    Array b = arrayCreate(c.count, true);

    // Benchmark
    hipEvent_t tstart, tstop;
    hipEventCreate(&tstart);
    hipEventCreate(&tstop);

    hipEventRecord(tstart); // Timing start
    for (size_t i = 0; i < num_iters; ++i)
        kernel<<<c.bpg, c.tpb, c.smem>>>(c.halo, a, b);
    hipEventRecord(tstop); // Timing stop
    hipEventSynchronize(tstop);
    ERRCHK_CUDA_KERNEL_ALWAYS();

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, tstart, tstop);
    hipEventDestroy(tstart);
    hipEventDestroy(tstop);

    const size_t bytes     = num_iters * sizeof(a.data[0]) * (a.count + b.count - 2 * c.halo);
    const double seconds   = (double)milliseconds / 1e3;
    const double bandwidth = bytes / seconds;
    printf("Effective bandwidth: %g GiB/s\n", bandwidth / pow(1024, 3));
    printf("\tBytes transferred: %g GiB\n", bytes / pow(1024, 3));
    printf("\tTime elapsed: %g ms\n", (double)milliseconds);

    // File
    const char* benchmark_dir = "microbenchmark.csv";
    FILE* fp                  = fopen(benchmark_dir, "a");
    ERRCHK_ALWAYS(fp);
    ERRCHK_ALWAYS(fp);
    // format
    // 'usesmem, maxthreadsperblock, problemsize, workingsetsize, milliseconds, effectivebandwidth'
    fprintf(fp, "%d,%d,%lu,%lu,%g,%g\n", USE_SMEM, MAX_THREADS_PER_BLOCK, c.count * sizeof(double),
            (2 * c.halo + 1) * sizeof(double), (double)milliseconds, bandwidth);
    fclose(fp);

    // Free
    arrayDestroy(&a);
    arrayDestroy(&b);
}

void
printDeviceInfo(const int device_id)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device_id);
    printf("--------------------------------------------------\n");
    printf("Device Number: %d\n", device_id);
    const size_t bus_id_max_len = 128;
    char bus_id[bus_id_max_len];
    hipDeviceGetPCIBusId(bus_id, bus_id_max_len, device_id);
    printf("  PCI bus ID: %s\n", bus_id);
    printf("    Device name: %s\n", props.name);
    printf("    Compute capability: %d.%d\n", props.major, props.minor);

    // Compute
    printf("  Compute\n");
    printf("    Clock rate (GHz): %g\n", props.clockRate / 1e6); // KHz -> GHz
    printf("    Stream processors: %d\n", props.multiProcessorCount);
    printf(
        "    Compute mode: %d\n",
        (int)props
            .computeMode); // https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g7eb25f5413a962faad0956d92bae10d0
    // Memory
    printf("  Global memory\n");
    printf("    Memory Clock Rate (MHz): %d\n", props.memoryClockRate / (1000));
    printf("    Memory Bus Width (bits): %d\n", props.memoryBusWidth);
    printf("    Peak Memory Bandwidth (GiB/s): %f\n",
           2 * (props.memoryClockRate * 1e3) * props.memoryBusWidth / (8. * 1024. * 1024. * 1024.));
    printf("    ECC enabled: %d\n", props.ECCEnabled);

    // Memory usage
    size_t free_bytes, total_bytes;
    hipMemGetInfo(&free_bytes, &total_bytes);
    const size_t used_bytes = total_bytes - free_bytes;
    printf("    Total global mem: %.2f GiB\n", props.totalGlobalMem / (1024.0 * 1024 * 1024));
    printf("    Gmem used (GiB): %.2f\n", used_bytes / (1024.0 * 1024 * 1024));
    printf("    Gmem memory free (GiB): %.2f\n", free_bytes / (1024.0 * 1024 * 1024));
    printf("    Gmem memory total (GiB): %.2f\n", total_bytes / (1024.0 * 1024 * 1024));
    printf("  Caches\n");
#if !AC_USE_HIP
    printf("    Local L1 cache supported: %d\n", props.localL1CacheSupported);
    printf("    Global L1 cache supported: %d\n", props.globalL1CacheSupported);
#endif
    printf("    L2 size: %d KiB\n", props.l2CacheSize / (1024));
    printf("    Shared memory per block: %lu\n", props.sharedMemPerBlock);
    printf("  Other\n");
    printf("    Warp size: %d\n", props.warpSize);
    printf("--------------------------------------------------\n");
}

int
main(int argc, char* argv[])
{
    hipProfilerStop();
    if (argc != 3) {
        fprintf(stderr, "Usage: ./benchmark <problem size> <working set size>\n");
        fprintf(stderr, "       ./benchmark 0 0 # To use the defaults\n");
        return EXIT_FAILURE;
    }
    const size_t arg0 = (size_t)atol(argv[1]);
    const size_t arg1 = (size_t)atol(argv[2]);

    const size_t problem_size     = arg0 ? arg0 : 268435456; // 256 MiB default
    const size_t working_set_size = arg1 ? arg1 : 8;         // 8 byte default (r=0)
    const int halo                = ((working_set_size / sizeof(double)) - 1) / 2;
    const size_t count            = problem_size / sizeof(double);
    ERRCHK(working_set_size <= problem_size);

    if (working_set_size > problem_size) {
        fprintf(stderr, "Invalid working set size: %lu > %lu\n", working_set_size, problem_size);
        return EXIT_FAILURE;
    }

    printDeviceInfo(0);
    printf("USE_SMEM=%d\n", USE_SMEM);
    printf("MAX_THREADS_PER_BLOCK=%d\n", MAX_THREADS_PER_BLOCK);

    // hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
    // hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    KernelConfig c = autotune(count, halo);
    verify(c);
    hipProfilerStart();
    benchmark(c);
    hipProfilerStop();
    return EXIT_SUCCESS;
}